#include "hip/hip_runtime.h"
#include "../lib/gpu_math.h"
#define CUDA_NUM_THREADS 256
#define CUDA_NUM_BLOCKS 48
#define warpSize 32

// device functions for warp-based reduction using shufl operations
// TODO: should probably just be factored out into gpu_math or gpu_util
template<class T>
__device__   __forceinline__ T warp_sum(T mySum) {
  for (int offset = warpSize >> 1; offset > 0; offset >>= 1)
    mySum += shuffle_down(mySum, offset);
  return mySum;
}

__device__ __forceinline__
bool isNotDiv32(unsigned int val) {
  return val & 31;
}

/* requires blockDim.x <= 1024, blockDim.y == 1 */
template<class T>
__device__   __forceinline__ T block_sum(T mySum) {
  const unsigned int lane = threadIdx.x & 31;
  const unsigned int wid = threadIdx.x >> 5;

  __shared__ T scratch[32];

  mySum = warp_sum(mySum);
  if (lane == 0) scratch[wid] = mySum;
  __syncthreads();

  if (wid == 0) {
    mySum = (threadIdx.x < blockDim.x >> 5) ? scratch[lane] : 0;
    mySum = warp_sum(mySum);
    if (threadIdx.x == 0 && isNotDiv32(blockDim.x))
      mySum += scratch[blockDim.x >> 5];
  }
  return mySum;
}

__global__
void gpu_l2(const float* optgrid, const float* screengrid, float* scoregrid,
    size_t gsize) {
  unsigned tidx = blockDim.x * blockIdx.x + threadIdx.x;
  unsigned nthreads = blockDim.x * gridDim.x;
  // optimized grids
  float sum = 0.;
  for (size_t k=tidx; k<gsize; k+=nthreads) {
    float diff = optgrid[k] - screengrid[k];
    float sqdiff = diff * diff;
    sum += sqdiff;
  }
  float total = block_sum<float>(sum);
  if (tidx == 0)
    *scoregrid = sqrtf(total);
}

void do_gpu_l2(const float* optgrid, const float* screengrid, float* scoregrid,
    size_t gsize) {
  unsigned block_multiple = gsize / CUDA_NUM_THREADS;
  unsigned nblocks = block_multiple < CUDA_NUM_BLOCKS ? block_multiple : CUDA_NUM_BLOCKS;
  gpu_l2<<<nblocks, CUDA_NUM_THREADS>>>(optgrid, screengrid, scoregrid, gsize);
}
