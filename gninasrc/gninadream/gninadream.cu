#include "hip/hip_runtime.h"
#include "../lib/gpu_math.h"
#define CUDA_NUM_THREADS 256
#define warpSize 32

// device functions for warp-based reduction using shufl operations
// TODO: should probably just be factored out into gpu_math or gpu_util
template<class T>
__device__   __forceinline__ T warp_sum(T mySum) {
  for (int offset = warpSize >> 1; offset > 0; offset >>= 1)
    mySum += shuffle_down(mySum, offset);
  return mySum;
}

__device__ __forceinline__
bool isNotDiv32(unsigned int val) {
  return val & 31;
}

/* requires blockDim.x <= 1024, blockDim.y == 1 */
template<class T>
__device__   __forceinline__ T block_sum(T mySum) {
  const unsigned int lane = threadIdx.x & 31;
  const unsigned int wid = threadIdx.x >> 5;

  __shared__ T scratch[32];

  mySum = warp_sum(mySum);
  if (lane == 0) scratch[wid] = mySum;
  __syncthreads();

  if (wid == 0) {
    mySum = (threadIdx.x < blockDim.x >> 5) ? scratch[lane] : 0;
    mySum = warp_sum(mySum);
    if (threadIdx.x == 0 && isNotDiv32(blockDim.x))
      mySum += scratch[blockDim.x >> 5];
  }
  return mySum;
}

__global__
void gpu_l2(const float* optgrid, const float* screengrid, float* scoregrid,
    size_t gsize) {
  unsigned tidx = blockDim.x * blockIdx.x + threadIdx.x;
  unsigned nthreads = blockDim.x * gridDim.x;
  // optimized grids
  float sum = 0.;
  for (size_t k=0; k<gsize; k+=nthreads) {
    float diff = optgrid[k + tidx] - screengrid[k + tidx];
    float sqdiff = diff * diff;
    sum += sqdiff;
  }
  float total = block_sum<float>(sum);
  if (tidx == 0)
    *scoregrid = sqrtf(total);
}
