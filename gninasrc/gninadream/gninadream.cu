#include "hip/hip_runtime.h"
#include "../lib/gpu_math.h"
#define CUDA_NUM_THREADS 512
#define warpSize 32

// device functions for warp-based reduction using shufl operations
// TODO: should probably just be factored out into gpu_math or gpu_util
template<class T>
__device__   __forceinline__ T warp_sum(T mySum) {
  for (int offset = warpSize >> 1; offset > 0; offset >>= 1)
    mySum += shuffle_down(mySum, offset);
  return mySum;
}

__device__ __forceinline__
bool isNotDiv32(unsigned int val) {
  return val & 31;
}

/* requires blockDim.x <= 1024, blockDim.y == 1 */
template<class T>
__device__   __forceinline__ T block_sum(T mySum) {
  const unsigned int lane = threadIdx.x & 31;
  const unsigned int wid = threadIdx.x >> 5;

  __shared__ T scratch[32];

  mySum = warp_sum(mySum);
  if (lane == 0) scratch[wid] = mySum;
  __syncthreads();

  if (wid == 0) {
    mySum = (threadIdx.x < blockDim.x >> 5) ? scratch[lane] : 0;
    mySum = warp_sum(mySum);
    if (threadIdx.x == 0 && isNotDiv32(blockDim.x))
      mySum += scratch[blockDim.x >> 5];
  }
  return mySum;
}

__global__
void gpu_l2(const float* optgrid, const float* screengrid, float* scoregrid, 
    size_t M, size_t N, size_t gsize) {
  unsigned tidx = threadIdx.x;
  // optimized grids
  for (size_t i=0; i<M; ++i) {
    // conformers to screen against
    for (size_t j=0; j<N; ++j) {
      float sum = 0.;
      for (size_t k=0; k<gsize; k+=CUDA_NUM_THREADS) {
        float diff = optgrid[i * gsize + k] - screengrid[j * gsize + k];
        float sqdiff = diff * diff;
        sum += sqdiff;
      }
    float total = block_sum<float>(sum);
    if (tidx == 0)
      scoregrid[i * N + j] = sqrtf(total);
    }
  }
}
