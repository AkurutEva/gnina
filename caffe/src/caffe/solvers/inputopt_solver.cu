#include "hip/hip_runtime.h"
#include "caffe/sgd_solvers.hpp"
#define CUDA_NUM_THREADS 256
#define CUDA_NUM_BLOCKS 48

namespace caffe {

  template <typename Dtype>
  __global__ void threshold_blob(Dtype* offset_tblob, size_t blobsize, 
      Dtype threshold_value) {
    unsigned tidx = blockDim.x * blockIdx.x + threadIdx.x;
    unsigned nthreads = blockDim.x * gridDim.x;
    if (threshold_value) {
      Dtype clip_min = threshold_value > 0 ? -threshold_value : threshold_value;
      Dtype clip_max = threshold_value > 0 ? threshold_value : -threshold_value;
      for (size_t k=tidx; k<blobsize; k+=nthreads) {
        *(offset_tblob+k) = fmaxf(clip_min, fminf(*(offset_tblob+k), clip_max));
      }
    }
    else {
      for (size_t k=tidx; k<blobsize; k+=nthreads) {
        if (*(offset_tblob+k) < 0) {
          *(offset_tblob+k) = 0;
        }
      }
    }
  }

  template <typename Dtype>
  void InputOptSolver<Dtype>::DoThresholdGPU(Dtype* offset_tblob, size_t blobsize, 
      Dtype threshold_value) {
    unsigned block_multiple = blobsize / CUDA_NUM_THREADS;
    unsigned nblocks = block_multiple < CUDA_NUM_BLOCKS ? block_multiple : CUDA_NUM_BLOCKS;
    threshold_blob<<<nblocks, CUDA_NUM_THREADS>>>(offset_tblob, blobsize,
        threshold_value);
  }

  template void InputOptSolver<float>::DoThresholdGPU(float*, size_t, float);
  template void InputOptSolver<double>::DoThresholdGPU(double*, size_t, double);
}
